#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Neurons.h"

__global__ void verify_fired(Neuron *a, long *b, long *i){
  int ID = blockIdx.x;
  if (ID<NEURONS_X * NEURONS_Y){
    if(a[ID].I >= 30) b[i++] = ID;
  }
}
